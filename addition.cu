
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1000000 // Size of vectors

__global__ void vectorAdd(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int main() {
    int *a, *b, *c; // Host vectors
    int *d_a, *d_b, *d_c; // Device vectors

    // Allocate memory on host
    a = (int*)malloc(N * sizeof(int));
    b = (int*)malloc(N * sizeof(int));
    c = (int*)malloc(N * sizeof(int));

    // Allocate memory on device
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_c, N * sizeof(int));

    // Initialize vectors on host
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i;
    }

    // Copy vectors from host to device
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block sizes
    int blockSize = 256;
    int gridSize = (int)ceil((float)N/blockSize);

    // Launch kernel
    vectorAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Verify results
    for (int i = 0; i < 10; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Free memory
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}

